#include "hip/hip_runtime.h"
/*
MIT License

Copyright (c) 2024 Marco Barbone

Permission is hereby granted, free of charge, to any person obtaining a copy
of this software and associated documentation files (the "Software"), to deal
in the Software without restriction, including without limitation the rights
to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
copies of the Software, and to permit persons to whom the Software is
furnished to do so, subject to the following conditions:

The above copyright notice and this permission notice shall be included in all
copies or substantial portions of the Software.

THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
SOFTWARE.
*/

#include <philox/philox.h>

#include <catch2/catch_all.hpp>

static inline constexpr auto TESTS = 1 << 15;

template <typename T>
__global__ void testSeeding(std::uint64_t seed) {
    T rng(seed, 0, 0);
}

template <typename T>
__global__ void testGeneration(std::uint64_t seed, std::uint64_t* out) {
    T rng(seed, 0, 0);
    for (auto i = 0; i < TESTS; ++i) { out[i] = rng(); }
}

TEST_CASE("Philox4 32 INIT") {
    const auto seed = std::random_device()();
    INFO("SEED: " << seed);

    testSeeding<Philox4x32_10><<<1, 1>>>(seed);
    REQUIRE(hipDeviceSynchronize() == hipSuccess);
}

TEST_CASE("Philox2 64 INIT") {
    const auto seed = std::random_device()();
    INFO("SEED: " << seed);

    testSeeding<Philox2x64_10><<<1, 1>>>(seed);
    REQUIRE(hipDeviceSynchronize() == hipSuccess);
}

TEST_CASE("Philox4 32 INIT  GENERATE") {
    const auto seed = std::random_device()();
    INFO("SEED: " << seed);
    Philox4x32_10  reference(seed, 0, 0);
    std::uint64_t* out;

    REQUIRE(hipMallocManaged(&out, TESTS * sizeof(std::uint64_t)) == hipSuccess);
    REQUIRE(hipDeviceSynchronize() == hipSuccess);

    testGeneration<Philox4x32_10><<<1, 1>>>(seed, out);
    REQUIRE(hipDeviceSynchronize() == hipSuccess);

    for (auto i = 0; i < TESTS; ++i) { REQUIRE(out[i] == reference()); }
    REQUIRE(hipFree(out) == hipSuccess);
}

TEST_CASE("Philox2 64 INIT GENERATE") {
    const auto seed = std::random_device()();
    INFO("SEED: " << seed);
    Philox2x64_10  reference(seed, 0, 0);
    std::uint64_t* out;

    REQUIRE(hipMallocManaged(&out, TESTS * sizeof(std::uint64_t)) == hipSuccess);
    REQUIRE(hipDeviceSynchronize() == hipSuccess);

    testGeneration<Philox2x64_10><<<1, 1>>>(seed, out);
    REQUIRE(hipDeviceSynchronize() == hipSuccess);

    for (auto i = 0; i < TESTS; ++i) { REQUIRE(out[i] == reference()); }

    REQUIRE(hipFree(out) == hipSuccess);
}
